#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1024  // 定义矩阵的大小
#define TILE_WIDTH 16  // 定义共享内存块的大小

__global__ void matrixMulSharedKernel(float *A, float *B, float *C, int width) {
    __shared__ float shared_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float shared_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float value = 0.0f;

    // 分块加载矩阵到共享内存
    for (int m = 0; m < (width + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        if (row < width && m * TILE_WIDTH + tx < width)
            shared_A[ty][tx] = A[row * width + m * TILE_WIDTH + tx];
        else
            shared_A[ty][tx] = 0.0;

        if (col < width && m * TILE_WIDTH + ty < width)
            shared_B[ty][tx] = B[(m * TILE_WIDTH + ty) * width + col];
        else
            shared_B[ty][tx] = 0.0;

        __syncthreads();

        // 计算当前块的矩阵乘积
        for (int k = 0; k < TILE_WIDTH; ++k) {
            value += shared_A[ty][k] * shared_B[k][tx];
        }

        __syncthreads();
    }

    if (row < width && col < width) {
        C[row * width + col] = value;
    }
}

void matrixMultiplicationShared(float *h_A, float *h_B, float *h_C, int width) {
    int size = width * width * sizeof(float);
    float *d_A, *d_B, *d_C;

    // 分配设备内存
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 拷贝数据到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义线程块和网格
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize((width + TILE_WIDTH - 1) / TILE_WIDTH, (width + TILE_WIDTH - 1) / TILE_WIDTH);

    // 启动核函数
    matrixMulSharedKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, width);

    // 拷贝结果回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int width = N;
    int size = width * width;
    float *h_A = new float[size];
    float *h_B = new float[size];
    float *h_C = new float[size];

    // 初始化矩阵
    for (int i = 0; i < size; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // 计时优化后的矩阵乘法
    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplicationShared(h_A, h_B, h_C, width);
    auto stop = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> duration = stop - start;
    std::cout << "Time after optimization: " << duration.count() << " ms" << std::endl;

    // 清理
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}