#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

#define N 1024  // 定义矩阵的大小

__global__ void matrixMulKernel(float *A, float *B, float *C, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < width && col < width) {
        float value = 0.0f;
        for (int k = 0; k < width; k++) {
            value += A[row * width + k] * B[k * width + col];
        }
        C[row * width + col] = value;
    }
}

void matrixMultiplication(float *h_A, float *h_B, float *h_C, int width) {
    int size = width * width * sizeof(float);
    float *d_A, *d_B, *d_C;

    // 分配设备内存
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 拷贝数据到设备
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义线程块和网格
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (width + blockSize.y - 1) / blockSize.y);

    // 启动核函数
    matrixMulKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, width);

    // 拷贝结果回主机
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int width = N;
    int size = width * width;
    float *h_A = new float[size];
    float *h_B = new float[size];
    float *h_C = new float[size];

    // 初始化矩阵
    for (int i = 0; i < size; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 1.0f;
    }

    // 计时优化前的矩阵乘法
    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiplication(h_A, h_B, h_C, width);
    auto stop = std::chrono::high_resolution_clock::now();

    std::chrono::duration<float, std::milli> duration = stop - start;
    std::cout << "Time before optimization: " << duration.count() << " ms" << std::endl;

    // 清理
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}